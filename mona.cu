#include "hip/hip_runtime.h"
// written by nick welch <nick@incise.org>.  author disclaims copyright.

#ifndef NUM_POINTS
#define NUM_POINTS 6
#endif

#ifndef NUM_SHAPES
#define NUM_SHAPES 40
#endif

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <unistd.h>
#include <limits.h>

#include <cairo.h>
#include <cairo-xlib.h>

#include <hip/hip_runtime.h>

#define RANDINT(max) (int)((random() / (double)RAND_MAX) * (max))
#define RANDDOUBLE(max) ((random() / (double)RAND_MAX) * max)
#define ABS(val) ((val) < 0 ? -(val) : (val))
#define CLAMP(val, min, max) ((val) < (min) ? (min) : \
                              (val) > (max) ? (max) : (val))

//The CUDA block size
#define BLOCK_SIZE 16

int WIDTH;
int HEIGHT;

//////////////////////// X11 stuff ////////////////////////
#ifdef SHOWWINDOW

#include <X11/Xlib.h>

Display * dpy;
int screen;
Window win;
GC gc;
Pixmap pixmap;

void x_init(void)
{
    if(!(dpy = XOpenDisplay(NULL)))
    {
        fprintf(stderr, "Failed to open X display %s\n", XDisplayName(NULL));
        exit(1);
    }

    screen = DefaultScreen(dpy);

    XSetWindowAttributes attr;
    attr.background_pixmap = ParentRelative;
    win = XCreateWindow(dpy, DefaultRootWindow(dpy), 0, 0,
                   WIDTH, HEIGHT, 0,
                   DefaultDepth(dpy, screen), CopyFromParent, DefaultVisual(dpy, screen),
                   CWBackPixmap, &attr);

    pixmap = XCreatePixmap(dpy, win, WIDTH, HEIGHT,
            DefaultDepth(dpy, screen));

    gc = XCreateGC(dpy, pixmap, 0, NULL);

    XSelectInput(dpy, win, ExposureMask);

    XMapWindow(dpy, win);
}
#endif
//////////////////////// end X11 stuff ////////////////////////

typedef struct {
    double x, y;
} point_t;

typedef struct {
    double r, g, b, a;
    point_t points[NUM_POINTS];
} shape_t;

shape_t dna_best[NUM_SHAPES];
shape_t dna_test[NUM_SHAPES];

int mutated_shape;

void draw_shape(shape_t * dna, cairo_t * cr, int i)
{
    cairo_set_line_width(cr, 0);
    shape_t * shape = &dna[i];
    cairo_set_source_rgba(cr, shape->r, shape->g, shape->b, shape->a);
    cairo_move_to(cr, shape->points[0].x, shape->points[0].y);
    for(int j = 1; j < NUM_POINTS; j++)
        cairo_line_to(cr, shape->points[j].x, shape->points[j].y);
    cairo_fill(cr);
}

void draw_dna(shape_t * dna, cairo_t * cr)
{
    cairo_set_source_rgb(cr, 1, 1, 1);
    cairo_rectangle(cr, 0, 0, WIDTH, HEIGHT);
    cairo_fill(cr);
    for(int i = 0; i < NUM_SHAPES; i++)
        draw_shape(dna, cr, i);
}

void init_dna(shape_t * dna)
{
    for(int i = 0; i < NUM_SHAPES; i++)
    {
        for(int j = 0; j < NUM_POINTS; j++)
        {
            dna[i].points[j].x = RANDDOUBLE(WIDTH);
            dna[i].points[j].y = RANDDOUBLE(HEIGHT);
        }
        dna[i].r = RANDDOUBLE(1);
        dna[i].g = RANDDOUBLE(1);
        dna[i].b = RANDDOUBLE(1);
        dna[i].a = RANDDOUBLE(1);
        //dna[i].r = 0.5;
        //dna[i].g = 0.5;
        //dna[i].b = 0.5;
        //dna[i].a = 1;
    }
}

int mutate(void)
{
    mutated_shape = RANDINT(NUM_SHAPES);
    double roulette = RANDDOUBLE(2.8);
    double drastic = RANDDOUBLE(2);
     
    // mutate color
    if(roulette<1)
    {
        if(dna_test[mutated_shape].a < 0.01 // completely transparent shapes are stupid
                || roulette<0.25)
        {
            if(drastic < 1)
            {
                dna_test[mutated_shape].a += RANDDOUBLE(0.1);
                dna_test[mutated_shape].a = CLAMP(dna_test[mutated_shape].a, 0.0, 1.0);
            }
            else
                dna_test[mutated_shape].a = RANDDOUBLE(1.0);
        }
        else if(roulette<0.50)
        {
            if(drastic < 1)
            {
                dna_test[mutated_shape].r += RANDDOUBLE(0.1);
                dna_test[mutated_shape].r = CLAMP(dna_test[mutated_shape].r, 0.0, 1.0);
            }
            else
                dna_test[mutated_shape].r = RANDDOUBLE(1.0);
        }
        else if(roulette<0.75)
        {
            if(drastic < 1)
            {
                dna_test[mutated_shape].g += RANDDOUBLE(0.1);
                dna_test[mutated_shape].g = CLAMP(dna_test[mutated_shape].g, 0.0, 1.0);
            }
            else
                dna_test[mutated_shape].g = RANDDOUBLE(1.0);
        }
        else
        {
            if(drastic < 1)
            {
                dna_test[mutated_shape].b += RANDDOUBLE(0.1);
                dna_test[mutated_shape].b = CLAMP(dna_test[mutated_shape].b, 0.0, 1.0);
            }
            else
                dna_test[mutated_shape].b = RANDDOUBLE(1.0);
        }
    }
    
    // mutate shape
    else if(roulette < 2.0)
    {
        int point_i = RANDINT(NUM_POINTS);
        if(roulette<1.5)
        {
            if(drastic < 1)
            {
                dna_test[mutated_shape].points[point_i].x += (int)RANDDOUBLE(WIDTH/10.0);
                dna_test[mutated_shape].points[point_i].x = CLAMP(dna_test[mutated_shape].points[point_i].x, 0, WIDTH-1);
            }
            else
                dna_test[mutated_shape].points[point_i].x = RANDDOUBLE(WIDTH);
        }
        else
        {
            if(drastic < 1)
            {
                dna_test[mutated_shape].points[point_i].y += (int)RANDDOUBLE(HEIGHT/10.0);
                dna_test[mutated_shape].points[point_i].y = CLAMP(dna_test[mutated_shape].points[point_i].y, 0, HEIGHT-1);
            }
            else
                dna_test[mutated_shape].points[point_i].y = RANDDOUBLE(HEIGHT);
        }
    }

    // mutate stacking
    else
    {
        int destination = RANDINT(NUM_SHAPES);
        shape_t s = dna_test[mutated_shape];
        dna_test[mutated_shape] = dna_test[destination];
        dna_test[destination] = s;
        return destination;
    }
    return -1;

}

int MAX_FITNESS = -1;

unsigned char * goal_data = NULL;

void differenceKernel(unsigned char * test_data, unsigned char * goal_data, int * difference, int * my_max_fitness) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tx * HEIGHT + ty;

    int difference_s = 0;
    int my_max_fitness_s = 0;

    if(i < HEIGHT*WIDTH) {
	    int thispixel = ty*WIDTH*4 + tx*4;

	    unsigned char test_a = test_data[thispixel];
	    unsigned char test_r = test_data[thispixel + 1];
	    unsigned char test_g = test_data[thispixel + 2];
	    unsigned char test_b = test_data[thispixel + 3];

	    unsigned char goal_a = goal_data[thispixel];
	    unsigned char goal_r = goal_data[thispixel + 1];
	    unsigned char goal_g = goal_data[thispixel + 2];
	    unsigned char goal_b = goal_data[thispixel + 3];

	    if(MAX_FITNESS == -1)
		my_max_fitness_s += goal_a + goal_r + goal_g + goal_b;

	    difference_s += (ABS(test_a - goal_a) + ABS(test_r - goal_r) + ABS(test_g - goal_g) + ABS(test_b - goal_b));
    }

    my_max_fitness[i] = my_max_fitness_s;
    difference[i] = difference_s;
}

int difference(cairo_surface_t * test_surf, cairo_surface_t * goal_surf)
{
    unsigned char * test_data = cairo_image_surface_get_data(test_surf);
    if(!goal_data)
        goal_data = cairo_image_surface_get_data(goal_surf);

    unsigned char * test_data_d;
    unsigned char * goal_data_d;
    int * difference;
    int * my_max_fitness;
    int * difference_d;
    int * my_max_fitness_d;
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gridDim(ceil((float)WIDTH/(float)BLOCK_SIZE), ceil((float)HEIGHT/(float)BLOCK_SIZE), 1);

    //TODO: Make these pointers global and only malloc once during the entire program
    hipMalloc((void **)&test_data_d, sizeof(unsigned char)*WIDTH*HEIGHT);
    hipMalloc((void **)&goal_data_d, sizeof(unsigned char)*WIDTH*HEIGHT);
    hipMalloc((void **)&difference_d, sizeof(int)*WIDTH*HEIGHT);
    hipMalloc((void **)&my_max_fitness_d, sizeof(int)*WIDTH*HEIGHT);
    difference = (int *)malloc(sizeof(int)*WIDTH*HEIGHT);
    my_max_fitness = (int *)malloc(sizeof(int)*WIDTH*HEIGHT);

    //This will really slow things down. PCI-E bus will be a bottleneck.
    hipMemcpy(test_data_d, test_data, sizeof(unsigned char)*WIDTH*HEIGHT, hipMemcpyHostToDevice);
    hipMemcpy(goal_data_d, goal_data, sizeof(unsigned char)*WIDTH*HEIGHT, hipMemcpyHostToDevice);

    //Launch the kernel to compute the difference
    differenceKernel<<<gridDim, blockDim>>>(test_data_d, goal_data_d, difference_d, my_max_fitness_d);

    //Copy results from the device, another PCI-E bottleneck
    hipMemcpy(difference, difference_d, sizeof(int)*WIDTH*HEIGHT, hipMemcpyDeviceToHost);
    hipMemcpy(my_max_fitness, my_max_fitness_d, sizeof(int)*WIDTH*HEIGHT, hipMemcpyDeviceToHost);

    /*
    int difference = 0;

    int my_max_fitness = 0;

    #pragma omp parallel for 
    for(int y = 0; y < HEIGHT; y++)
    {
        for(int x = 0; x < WIDTH; x++)
        {
            int thispixel = y*WIDTH*4 + x*4;

            unsigned char test_a = test_data[thispixel];
            unsigned char test_r = test_data[thispixel + 1];
            unsigned char test_g = test_data[thispixel + 2];
            unsigned char test_b = test_data[thispixel + 3];

            unsigned char goal_a = goal_data[thispixel];
            unsigned char goal_r = goal_data[thispixel + 1];
            unsigned char goal_g = goal_data[thispixel + 2];
            unsigned char goal_b = goal_data[thispixel + 3];

            if(MAX_FITNESS == -1)
                my_max_fitness += goal_a + goal_r + goal_g + goal_b;

		#pragma omp atomic
	    difference += (ABS(test_a - goal_a) + ABS(test_r - goal_r) + ABS(test_g - goal_g) + ABS(test_b - goal_b));
        }
    }
    */

    //TODO: perform reduction on the GPU. Probalby won't be much speedup anyways
    int my_max_fitness_total = 0;
    int difference_total = 0;

    for(int i = 0; i < WIDTH*HEIGHT; i++) {
	    my_max_fitness_total += my_max_fitness[i];
	    difference_total += difference[i];
    }

    if(MAX_FITNESS == -1)
        MAX_FITNESS = my_max_fitness_total;

    hipFree(test_data_d);
    hipFree(goal_data_d);
    hipFree(difference_d);
    hipFree(my_max_fitness_d);
    free(difference);
    free(my_max_fitness);

    return difference_total;
}


void copy_surf_to(cairo_surface_t * surf, cairo_t * cr)
{
    cairo_set_source_rgb(cr, 1, 1, 1);
    cairo_rectangle(cr, 0, 0, WIDTH, HEIGHT);
    cairo_fill(cr);
    //cairo_set_operator(cr, CAIRO_OPERATOR_SOURCE);
    cairo_set_source_surface(cr, surf, 0, 0);
    cairo_paint(cr);
}

static void mainloop(cairo_surface_t * pngsurf)
{
    struct timeval start;
    gettimeofday(&start, NULL);

    init_dna(dna_best);
    memcpy((void *)dna_test, (const void *)dna_best, sizeof(shape_t) * NUM_SHAPES);

#ifdef SHOWWINDOW
    cairo_surface_t * xsurf = cairo_xlib_surface_create(
            dpy, pixmap, DefaultVisual(dpy, screen), WIDTH, HEIGHT);
    cairo_t * xcr = cairo_create(xsurf);
#endif

    cairo_surface_t * test_surf = cairo_image_surface_create(CAIRO_FORMAT_ARGB32, WIDTH, HEIGHT);
    cairo_t * test_cr = cairo_create(test_surf);

    cairo_surface_t * goalsurf = cairo_image_surface_create(CAIRO_FORMAT_ARGB32, WIDTH, HEIGHT);
    cairo_t * goalcr = cairo_create(goalsurf);
    copy_surf_to(pngsurf, goalcr);

    int lowestdiff = INT_MAX;
    int teststep = 0;
    int beststep = 0;
    for(;;) {
        int other_mutated = mutate();
        draw_dna(dna_test, test_cr);

        int diff = difference(test_surf, goalsurf);
        if(diff < lowestdiff)
        {
            beststep++;
            // test is good, copy to best
            dna_best[mutated_shape] = dna_test[mutated_shape];
            if(other_mutated >= 0)
                dna_best[other_mutated] = dna_test[other_mutated];
#ifdef SHOWWINDOW
            copy_surf_to(test_surf, xcr); // also copy to display
            XCopyArea(dpy, pixmap, win, gc,
                    0, 0,
                    WIDTH, HEIGHT,
                    0, 0);
#endif
            lowestdiff = diff;
        }
        else
        {
            // test sucks, copy best back over test
            dna_test[mutated_shape] = dna_best[mutated_shape];
            if(other_mutated >= 0)
                dna_test[other_mutated] = dna_best[other_mutated];
        }

        teststep++;

#ifdef TIMELIMIT
        struct timeval t;
        gettimeofday(&t, NULL);
        if(t.tv_sec - start.tv_sec > TIMELIMIT)
        {
            printf("%0.6f\n", ((MAX_FITNESS-lowestdiff) / (float)MAX_FITNESS)*100);
#ifdef DUMP
            char filename[50];
            sprintf(filename, "%d.data", getpid());
            FILE * f = fopen(filename, "w");
            fwrite(dna_best, sizeof(shape_t), NUM_SHAPES, f);
            fclose(f);
#endif
            return;
        }
#else
        if(teststep % 100 == 0)
            printf("Step = %d/%d\nFitness = %0.6f%%\n",
                    beststep, teststep, ((MAX_FITNESS-lowestdiff) / (float)MAX_FITNESS)*100);
#endif

#ifdef SHOWWINDOW
        if(teststep % 100 == 0 && XPending(dpy))
        {
            XEvent xev;
            XNextEvent(dpy, &xev);
            switch(xev.type) {
                case Expose:
                    XCopyArea(dpy, pixmap, win, gc,
                            xev.xexpose.x, xev.xexpose.y,
                            xev.xexpose.width, xev.xexpose.height,
                            xev.xexpose.x, xev.xexpose.y);
            }
        }
#endif
    }
}

int main(int argc, char ** argv) {
    cairo_surface_t * pngsurf;
    if(argc == 1)
        pngsurf = cairo_image_surface_create_from_png("mona.png");
    else
        pngsurf = cairo_image_surface_create_from_png(argv[1]);

    WIDTH = cairo_image_surface_get_width(pngsurf);
    HEIGHT = cairo_image_surface_get_height(pngsurf);

    srandom(getpid() + time(NULL));
    x_init();
    mainloop(pngsurf);
}

